#include "hip/hip_runtime.h"
#include <qelapsedtimer.h>
#include "AdaptiveMedianFilterCuda.cuh"
#include "hip/hip_runtime.h"
#include ""

#define MAX_AREA_SIZE 7
#define MEDIAN_BUFFER_SIZE (MAX_AREA_SIZE * MAX_AREA_SIZE + 1)

__device__ void quickSort(unsigned char *arr, int left, int right) {
	int i = left, j = right;
	int tmp;
	int pivot = arr[(left + right) / 2];

	/* partition */
	while (i <= j) {
		while (arr[i] < pivot)
			i++;
		while (arr[j] > pivot)
			j--;
		if (i <= j) {
			tmp = arr[i];
			arr[i] = arr[j];
			arr[j] = tmp;
			i++;
			j--;
		}
	};

	/* recursion */
	if (left < j)
		quickSort(arr, left, j);
	if (i < right)
		quickSort(arr, i, right);
}

__global__ void filterKernel(unsigned char *imageData, unsigned char *filteredImageData, int bytesPerLine,
	unsigned char *medianBuffer)
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	int width = gridDim.x;
	int height = gridDim.y;

	bool processed = false;

	int pixelOffset = y * bytesPerLine + x;

	unsigned int pixel = imageData[pixelOffset];

	// ������� ����������� �����
	int n = 3;

	unsigned char *median = medianBuffer + ((y * width) + x) * MEDIAN_BUFFER_SIZE;

	//thrust::device_vector<unsigned char> median(MAX_AREA_SIZE * MAX_AREA_SIZE + 1, 255);

	//std::array<unsigned char, MAX_AREA_SIZE * MAX_AREA_SIZE + 1> median;
	
	while (!processed) {
		// ����������� �������� ������� � �����������
		double zMin = 255;
		// ������������ �������� ������� � �����������
		double zMax = 0;
		// ������� �������� �������
		double zMed = 0;

		// ������ ����������� � ���� �������
		int sDelta = (n - 1) / 2;

		int processedPixelCount = 0;

		// �������� �� ����������� �����, �������� �������� ����������
		for (int sx = x - sDelta; sx <= x + sDelta; sx++) {
			for (int sy = y - sDelta; sy <= y + sDelta; sy++) {
				if (sx < 0 || sy < 0 || sx >= width || sy >= height) {
					continue;
				}

				unsigned int currentPixel = imageData[sy * bytesPerLine + sx];

				if (currentPixel < zMin) {
					zMin = currentPixel;
				}

				if (currentPixel > zMax) {
					zMax = currentPixel;
				}

				median[processedPixelCount] = currentPixel;

				processedPixelCount++;
			}
		}

		quickSort(median, 0, processedPixelCount);

		zMed = median[processedPixelCount / 2];

		double a1 = zMed - zMin;
		double a2 = zMed - zMax;

		if (a1 > 0 && a2 < 0) {
			double b1 = pixel - zMin;
			double b2 = pixel - zMax;

			if (b1 > 0 && b2 < 0) {
				filteredImageData[pixelOffset] = pixel;
			}
			else {
				filteredImageData[pixelOffset] = zMed;
			}

			processed = true;
		}
		else {
			n += 2;
			if (n > 7) {
				filteredImageData[pixelOffset] = zMed;
				processed = true;
			}
		}
	}
}

bool AdaptiveMedianFilterCuda::init() {
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
	}

	cudaEnabled = cudaStatus == hipSuccess;

	return cudaEnabled;
}

bool AdaptiveMedianFilterCuda::enabled() {
	return cudaEnabled;
}

bool AdaptiveMedianFilterCuda::filterImageWithCuda(int width, int height,
						 unsigned char *imageData, unsigned char *filteredImageData,
						 int bytesPerLine, int bytesCount, qint64 *computeOnlyTimeout) {
	unsigned char *dev_imageData = 0;
	unsigned char *dev_filteredImageData = 0;
	unsigned char *dev_medianBuffer = 0;
	hipError_t cudaStatus;
	dim3 image(width, height);

	cudaStatus = hipMalloc((void**)&dev_imageData, bytesCount);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for image data failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_filteredImageData, bytesCount);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for filtered image data failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_medianBuffer, MEDIAN_BUFFER_SIZE * width * height *
		sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for median buffer failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_imageData, imageData, bytesCount, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy for image data to device failed!");
		goto Error;
	}

	QElapsedTimer computeOnlyTimer;
	computeOnlyTimer.start();
	filterKernel<<<image, 1>>>(dev_imageData, dev_filteredImageData, bytesPerLine, dev_medianBuffer);
	*computeOnlyTimeout = computeOnlyTimer.elapsed();

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "filterKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching filterKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(filteredImageData, dev_filteredImageData, bytesCount, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy for filtered image data from device failed!");
		goto Error;
	}

Error:
	hipFree(dev_imageData);
	hipFree(dev_filteredImageData);
	hipFree(dev_medianBuffer);

	return cudaStatus == hipSuccess;
}